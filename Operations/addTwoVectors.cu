
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // since we're asking for one more thread than elements in the arrays
    // we need to handle size to make sure we don't access beyond the end of the array
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

// add vectors in parallel
void addWithCuda(int* c, const int* a, const int* b, int size) {
    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;

    // allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // launch a kernel on the GPU with one thread for each element.
    // 2 is number of blocks and (size + 1) / 2 is the number of threads in a block
    // we add 1 to size because we're asking for one additional thread rather than having one thread responsible for two items
    addKernel<<<2, (size + 1) / 2>>>(dev_c, dev_a, dev_b, size);
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}

int main(int argc, char** argv) {
    const int arraySize = 5;
    const int a[arraySize] = {  1,  2,  3,  4,  5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    addWithCuda(c, a, b, arraySize);

    printf("{1, 2, 3, 4, 5} + {10, 20, 30, 40, 50} = {%d, %d, %d, %d, %d}\n", c[0], c[1], c[2], c[3], c[4]);

    hipDeviceReset();

    return 0;
}
 