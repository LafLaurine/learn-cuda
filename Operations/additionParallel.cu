
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

void random_ints(int* a, int n)
{
   int i;
   for (i = 0; i < n; ++i) {
       a[i] = rand() %5000;
   }
}

// indexing an array with one element per thread
// M threads per block, a unique index for each thread is given by threadIdx.x + blockIdx.x * M

__global__ void add(int *a, int *b, int *c, int n)
{
    //blockDim.x represents threads per block
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // as we need to avoid to go beyond the end of the arrays, we need to define the limit
    if (index < n)
        c[index] = a[index] + b[index];
}

int main(void)
{
    // host copies of a, b, c
    int *a, *b, *c; 
    // device copies of a, b, c
    int *d_a, *d_b, *d_c; 
    int size = N * sizeof(int);

    // we need to allocate memory on the GPU
    // allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // allocate space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); 
    random_ints(a, N);
    b = (int *)malloc(size); 
    random_ints(b, N);
    c = (int *)malloc(size);

    // copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // launch add() kernel on the GPU
    add<<<(N + THREADS_PER_BLOCK-1)/ THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

    // copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // don't forget to free the memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // check error 
    printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));

    return 0;
}