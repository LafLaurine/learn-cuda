#include "hip/hip_runtime.h"
#include "convolution.hpp"

#include <cassert>

namespace IMAC
{
    __device__ float cu_clampf(const float val, const float minVal, const float maxVal)
    {
        return min(maxVal, max(minVal, val));
    }

    __global__ void cu_ex1(const uint imgWidth, const uint imgHeight, const uint matSize, uchar4 *inputImg, float *matConv, uchar4 *output)
    {
        for (int h = threadIdx.y + blockDim.y * blockIdx.y; h < imgHeight; h += blockDim.y * gridDim.y)
        {
            for (int w = threadIdx.x + blockDim.x * blockIdx.x; w < imgWidth; w += blockDim.x * gridDim.x)
            {
                float3 sum = make_float3(0.f, 0.f, 0.f);
                // Apply convolution
                for (uint j = 0; j < matSize; ++j)
                {
                    for (uint i = 0; i < matSize; ++i)
                    {
                        int dX = w + i - matSize / 2;
                        int dY = h + j - matSize / 2;

                        // Handle borders
                        if (dX < 0)
                            dX = 0;

                        if (dX >= imgWidth)
                            dX = imgWidth - 1;

                        if (dY < 0)
                            dY = 0;

                        if (dY >= imgHeight)
                            dY = imgHeight - 1;

                        const int idMat = j * matSize + i;
                        const int idPixel = dY * imgWidth + dX;
                        sum.x += (float)inputImg[idPixel].x * matConv[idMat];
                        sum.y += (float)inputImg[idPixel].y * matConv[idMat];
                        sum.z += (float)inputImg[idPixel].z * matConv[idMat];
                    }
                }
                const int index = w + imgWidth * h;
                output[index].x = __float2uint_rd(cu_clampf(sum.x, 0.f, 255.f));
                output[index].y = __float2uint_rd(cu_clampf(sum.y, 0.f, 255.f));
                output[index].z = __float2uint_rd(cu_clampf(sum.z, 0.f, 255.f));
                output[index].w = 255;
            }
        }
    }

    void ex1(const std::vector<uchar4> &inputImg, const uint imgWidth, const uint imgHeight, const std::vector<float> &matConv,
             const uint matSize, std::vector<uchar4> &output)
    {
        // 3 arrays for GPU
        uchar4 *d_inputImg = nullptr;
        float *d_matConv = nullptr;
        uchar4 *d_output = nullptr;

        // Allocate arrays
        HANDLE_ERROR(hipMalloc(&d_inputImg, sizeof(uchar4) * inputImg.size()));
        HANDLE_ERROR(hipMalloc(&d_matConv, sizeof(float) * matConv.size()));
        HANDLE_ERROR(hipMalloc(&d_output, sizeof(uchar4) * inputImg.size()));

        // Copy data from host to device
        HANDLE_ERROR(hipMemcpy(d_inputImg, inputImg.data(), sizeof(uchar4) * inputImg.size(), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(d_matConv, matConv.data(), sizeof(float) * matConv.size(), hipMemcpyHostToDevice));

        // Launch kernel
        const uint BLOCK_SIZE = 32;
        dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight / threadsPerBlock.y));
        cu_ex1<<<numBlocks, threadsPerBlock>>>(imgWidth, imgHeight, matSize, d_inputImg, d_matConv, d_output);
        HANDLE_ERROR(hipDeviceSynchronize());

        // Copy data from device to host (output array)
        HANDLE_ERROR(hipMemcpy(output.data(), d_output, sizeof(uchar4) * inputImg.size(), hipMemcpyDeviceToHost));

        // Free arrays on device
        HANDLE_ERROR(hipFree(d_inputImg));
        HANDLE_ERROR(hipFree(d_matConv));
        HANDLE_ERROR(hipFree(d_output));
    }

    // Constant memory
    __constant__ float MAT_CONV[225]; // it is matConv.size()

    __global__ void cu_ex2(const uint imgWidth, const uint imgHeight, const uint matSize, uchar4 *inputImg, uchar4 *output)
    {
        for (int h = threadIdx.y + blockDim.y * blockIdx.y; h < imgHeight; h += blockDim.y * gridDim.y)
        {
            for (int w = threadIdx.x + blockDim.x * blockIdx.x; w < imgWidth; w += blockDim.x * gridDim.x)
            {
                float3 sum = make_float3(0.f, 0.f, 0.f);
                // Apply convolution
                for (uint j = 0; j < matSize; ++j)
                {
                    for (uint i = 0; i < matSize; ++i)
                    {
                        int dX = w + i - matSize / 2;
                        int dY = h + j - matSize / 2;

                        // Handle borders
                        if (dX < 0)
                            dX = 0;

                        if (dX >= imgWidth)
                            dX = imgWidth - 1;

                        if (dY < 0)
                            dY = 0;

                        if (dY >= imgHeight)
                            dY = imgHeight - 1;

                        const int idMat = j * matSize + i;
                        const int idPixel = dY * imgWidth + dX;
                        sum.x += (float)inputImg[idPixel].x * MAT_CONV[idMat];
                        sum.y += (float)inputImg[idPixel].y * MAT_CONV[idMat];
                        sum.z += (float)inputImg[idPixel].z * MAT_CONV[idMat];
                    }
                }
                const int index = w + imgWidth * h;
                output[index].x = __float2uint_rd(cu_clampf(sum.x, 0.f, 255.f));
                output[index].y = __float2uint_rd(cu_clampf(sum.y, 0.f, 255.f));
                output[index].z = __float2uint_rd(cu_clampf(sum.z, 0.f, 255.f));
                output[index].w = 255;
            }
        }
    }

    void ex2(const std::vector<uchar4> &inputImg, const uint imgWidth, const uint imgHeight, const std::vector<float> &matConv,
             const uint matSize, std::vector<uchar4> &output)
    {
        assert(matConv.size() <= 225 && "matSize is too large for constant memory definition");

        // 2 arrays for GPU
        uchar4 *d_inputImg = nullptr;
        uchar4 *d_output = nullptr;

        // Allocate arrays
        HANDLE_ERROR(hipMalloc(&d_inputImg, sizeof(uchar4) * inputImg.size()));
        HANDLE_ERROR(hipMalloc(&d_output, sizeof(uchar4) * inputImg.size()));

        // Copy data from host to device
        HANDLE_ERROR(hipMemcpy(d_inputImg, inputImg.data(), sizeof(uchar4) * inputImg.size(), hipMemcpyHostToDevice));

        // Copy kernel to constant memory
        HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(MAT_CONV), matConv.data(), matConv.size() * sizeof(float)));

        // Launch kernel
        const uint BLOCK_SIZE = 32;
        dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight / threadsPerBlock.y));
        cu_ex2<<<numBlocks, threadsPerBlock>>>(imgWidth, imgHeight, matSize, d_inputImg, d_output);
        HANDLE_ERROR(hipDeviceSynchronize());

        // Copy data from device to host (output array)
        HANDLE_ERROR(hipMemcpy(output.data(), d_output, sizeof(uchar4) * inputImg.size(), hipMemcpyDeviceToHost));

        // Free arrays on device
        HANDLE_ERROR(hipFree(d_inputImg));
        HANDLE_ERROR(hipFree(d_output));
    }

    texture<uchar4, hipTextureType1D, hipReadModeElementType> inputTex;

    __global__ void cu_ex3(const uint imgWidth, const uint imgHeight, const uint matSize, float *matConv, uchar4 *output)
    {
        for (int h = threadIdx.y + blockDim.y * blockIdx.y; h < imgHeight; h += blockDim.y * gridDim.y)
        {
            for (int w = threadIdx.x + blockDim.x * blockIdx.x; w < imgWidth; w += blockDim.x * gridDim.x)
            {
                float3 sum = make_float3(0.f, 0.f, 0.f);
                // Apply convolution
                for (uint j = 0; j < matSize; ++j)
                {
                    for (uint i = 0; i < matSize; ++i)
                    {
                        int dX = w + i - matSize / 2;
                        int dY = h + j - matSize / 2;

                        // Handle borders
                        if (dX < 0)
                            dX = 0;

                        if (dX >= imgWidth)
                            dX = imgWidth - 1;

                        if (dY < 0)
                            dY = 0;

                        if (dY >= imgHeight)
                            dY = imgHeight - 1;

                        const int idMat = j * matSize + i;
                        const int idPixel = dY * imgWidth + dX;
                        // tex1D is used for hipArray. tex1Dfetch is for linear memory
                        uchar4 val = tex1Dfetch<uchar4>(inputTex, idPixel);
                        sum.x += (float)val.x * matConv[idMat];
                        sum.y += (float)val.y * matConv[idMat];
                        sum.z += (float)val.z * matConv[idMat];
                    }
                }
                const int index = w + imgWidth * h;
                output[index].x = __float2uint_rd(cu_clampf(sum.x, 0.f, 255.f));
                output[index].y = __float2uint_rd(cu_clampf(sum.y, 0.f, 255.f));
                output[index].z = __float2uint_rd(cu_clampf(sum.z, 0.f, 255.f));
                output[index].w = 255;
            }
        }
    }

    void ex3(const std::vector<uchar4> &inputImg, const uint imgWidth, const uint imgHeight, const std::vector<float> &matConv,
             const uint matSize, std::vector<uchar4> &output)
    {
        // 3 arrays for GPU
        uchar4 *d_inputImg = nullptr;
        float *d_matConv = nullptr;
        uchar4 *d_output = nullptr;

        // Allocate arrays
        HANDLE_ERROR(hipMalloc(&d_inputImg, sizeof(uchar4) * inputImg.size()));
        HANDLE_ERROR(hipMalloc(&d_matConv, sizeof(float) * matConv.size()));
        HANDLE_ERROR(hipMalloc(&d_output, sizeof(uchar4) * inputImg.size()));

        // Copy data from host to device
        HANDLE_ERROR(hipMemcpy(d_inputImg, inputImg.data(), sizeof(uchar4) * inputImg.size(), hipMemcpyHostToDevice));
        HANDLE_ERROR(hipMemcpy(d_matConv, matConv.data(), sizeof(float) * matConv.size(), hipMemcpyHostToDevice));

        // Bind input image to texture memory
        HANDLE_ERROR(hipBindTexture(0, inputTex, d_inputImg, sizeof(uchar4) * inputImg.size()));

        // Launch kernel
        const uint BLOCK_SIZE = 32;
        dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight / threadsPerBlock.y));
        cu_ex3<<<numBlocks, threadsPerBlock>>>(imgWidth, imgHeight, matSize, d_matConv, d_output);
        HANDLE_ERROR(hipDeviceSynchronize());

        // Copy data from device to host (output array)
        HANDLE_ERROR(hipMemcpy(output.data(), d_output, sizeof(uchar4) * inputImg.size(), hipMemcpyDeviceToHost));

        // Free arrays on device
        HANDLE_ERROR(hipFree(d_inputImg));
        HANDLE_ERROR(hipFree(d_matConv));
        HANDLE_ERROR(hipFree(d_output));
    }

    texture<uchar4, 2> inputTex2D;

    __global__ void cu_ex4(const uint imgWidth, const uint imgHeight, float *matConv, const uint matSize, uchar4 *output)
    {
        for (int h = threadIdx.y + blockDim.y * blockIdx.y; h < imgHeight; h += blockDim.y * gridDim.y)
        {
            for (int w = threadIdx.x + blockDim.x * blockIdx.x; w < imgWidth; w += blockDim.x * gridDim.x)
            {
                float3 sum = make_float3(0.f, 0.f, 0.f);
                // Apply convolution
                for (uint j = 0; j < matSize; ++j)
                {
                    for (uint i = 0; i < matSize; ++i)
                    {
                        int dX = w + i - matSize / 2;
                        int dY = h + j - matSize / 2;

                        // Handle borders
                        if (dX < 0)
                            dX = 0;

                        if (dX >= imgWidth)
                            dX = imgWidth - 1;

                        if (dY < 0)
                            dY = 0;

                        if (dY >= imgHeight)
                            dY = imgHeight - 1;

                        const int idMat = j * matSize + i;
                        uchar4 val = tex2D(inputTex2D, dX, dY);
                        sum.x += (float)val.x * matConv[idMat];
                        sum.y += (float)val.y * matConv[idMat];
                        sum.z += (float)val.z * matConv[idMat];
                    }
                }
                const int index = w + imgWidth * h;
                output[index].x = __float2uint_rd(cu_clampf(sum.x, 0.f, 255.f));
                output[index].y = __float2uint_rd(cu_clampf(sum.y, 0.f, 255.f));
                output[index].z = __float2uint_rd(cu_clampf(sum.z, 0.f, 255.f));
                output[index].w = 255;
            }
        }
    }

    void ex4(const std::vector<uchar4> &inputImg, const uint imgWidth, const uint imgHeight, const std::vector<float> &matConv,
             const uint matSize, std::vector<uchar4> &output)
    {
        // 3 arrays for GPU
        uchar4 *d_inputImg = nullptr;
        float *d_matConv = nullptr;
        uchar4 *d_output = nullptr;
        size_t pitch;

        // Allocate arrays
        HANDLE_ERROR(hipMallocPitch(&d_inputImg, &pitch, imgWidth * sizeof(uchar4), imgHeight));
        HANDLE_ERROR(hipMalloc(&d_matConv, sizeof(float) * matConv.size()));
        HANDLE_ERROR(hipMalloc(&d_output, sizeof(uchar4) * inputImg.size()));
        
        // Copy from host to device
        HANDLE_ERROR(hipMemcpy2D(d_inputImg, pitch, inputImg.data(), sizeof(uchar4) * imgWidth, imgWidth * sizeof(uchar4), imgHeight, hipMemcpyHostToDevice));

        HANDLE_ERROR(hipBindTexture2D(NULL, inputTex2D, d_inputImg, inputTex2D.channelDesc, imgWidth, imgHeight, pitch));

        // launch kernel
        const uint BLOCK_SIZE = 32;
        dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight / threadsPerBlock.y));
        cu_ex4<<<numBlocks, threadsPerBlock>>>(imgWidth, imgHeight, d_matConv, matSize, d_output);
        HANDLE_ERROR(hipDeviceSynchronize());

        // Copy data from device to host (output array)
        HANDLE_ERROR(hipMemcpy(output.data(), d_output, sizeof(uchar4) * inputImg.size(), hipMemcpyDeviceToHost));

        // Free arrays on device
        HANDLE_ERROR(hipUnbindTexture(inputTex2D));
        HANDLE_ERROR(hipFree(d_output));
        HANDLE_ERROR(hipFree(d_inputImg));
        HANDLE_ERROR(hipFree(d_matConv));
    }
}