#include "hip/hip_runtime.h"
/*
* TP 2 - Convolution d'images
* --------------------------
* Mémoire constante et textures
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"
#include "chronoGPU.hpp"
#include <algorithm>

#define BLOCK_SIZE 32
#define KERNEL_SIZE 16

namespace IMAC
{

// ================================================== For image comparison
	std::ostream &operator <<(std::ostream &os, const uchar4 &c)
	{
		os << "[" << uint(c.x) << "," << uint(c.y) << "," << uint(c.z) << "," << uint(c.w) << "]";  
    	return os; 
	}

	void compareImages(const std::vector<uchar4> &a, const std::vector<uchar4> &b)
	{
		bool error = false;
		if (a.size() != b.size())
		{
			std::cout << "Size is different !" << std::endl;
			error = true;
		}
		else
		{
			for (uint i = 0; i < a.size(); ++i)
			{
				// Floating precision can cause small difference between host and device
				if (std::abs(a[i].x - b[i].x) > 2 || std::abs(a[i].y - b[i].y) > 2 
					|| std::abs(a[i].z - b[i].z) > 2 || std::abs(a[i].w - b[i].w) > 2)
				{
					std::cout << "Error at index " << i << ": a = " << a[i] << " - b = " << b[i] << " - " << std::abs(a[i].x - b[i].x) << std::endl;
					error = true;
					break;
				}
			}
		}
		if (error)
		{
			std::cout << " -> You failed, retry!" << std::endl;
		}
		else
		{
			std::cout << " -> Well done!" << std::endl;
		}
	}

// ==================================================
	
	__global__ void applyConvolution(const unsigned char* dev_input, const uint imgWidth, const uint imgHeight, const uint matSize, float* dev_matConv, unsigned char* dev_output)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		int index = (idy * imgWidth + idx) * 4;
		if (idx < imgWidth && idy < imgHeight){
			float3 sum = make_float3(0.f,0.f,0.f);
			
			// Apply convolution
			for (uint j = 0; j < matSize; j++) 
			{
				for (uint i = 0; i < matSize; i++) 
				{
					int dX = idx + i - matSize / 2;
					int dY = idy + j - matSize / 2;

					// Handle borders
					if (dX < 0) 
						dX = 0;

					if (dX >= imgWidth) 
						dX = imgWidth - 1;

					if (dY < 0) 
						dY = 0;

					if (dY >= imgHeight) 
						dY = imgHeight - 1;
					
					const int idMat = j * matSize + i;
					const int idPixel = (dY * imgWidth + dX) * 4;
					sum.x += (float)dev_input[idPixel] * dev_matConv[idMat];
					sum.y += (float)dev_input[idPixel+1] * dev_matConv[idMat];
					sum.z += (float)dev_input[idPixel+2] * dev_matConv[idMat];
				}
			}
			dev_output[index] = (uchar)max(0.f,min(255.f,sum.x));
			dev_output[index+1] = (uchar)max(0.f,min(255.f,sum.y));
			dev_output[index+2] = (uchar)max(0.f,min(255.f,sum.z));
			dev_output[index+3] = 255;
		}
	}

	__device__ __constant__ float dev_matConv[KERNEL_SIZE * KERNEL_SIZE];

	__global__ void applyConvolutionv2(const unsigned char* dev_input, const uint imgWidth, const uint imgHeight,  const uint matSize, unsigned char* dev_output)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		int index = (idy * imgWidth + idx) * 4;
		if (idx < imgWidth && idy < imgHeight){
			float3 sum = make_float3(0.f,0.f,0.f);
			for (uint j = 0; j < matSize; j++) 
			{
				for (uint i = 0; i < matSize; i++) 
				{
					int dX = idx + i - matSize / 2;
					int dY = idy + j - matSize / 2;

					// Handle borders
					if (dX < 0) 
						dX = 0;

					if (dX >= imgWidth) 
						dX = imgWidth - 1;

					if (dY < 0) 
						dY = 0;

					if (dY >= imgHeight) 
						dY = imgHeight - 1;
					
					const int idMat = j * matSize + i;
					const int idPixel = (dY * imgWidth + dX) * 4;
					sum.x += (float)dev_input[idPixel] * dev_matConv[idMat];
					sum.y += (float)dev_input[idPixel+1] * dev_matConv[idMat];
					sum.z += (float)dev_input[idPixel+2] * dev_matConv[idMat];
				}
			}
			dev_output[index] = (uchar)max(0.f,min(255.f,sum.x));
			dev_output[index+1] = (uchar)max(0.f,min(255.f,sum.y));
			dev_output[index+2] = (uchar)max(0.f,min(255.f,sum.z));
			dev_output[index+3] = 255;
		}
	}

	texture<uchar,1,hipReadModeElementType> texRef;
	texture<uchar,2,hipReadModeElementType> tex2DRef;

	__global__ void applyConvolutionv3(const uint imgWidth, const uint imgHeight,  const uint matSize, unsigned char* dev_output)
	{
		unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
		int index = (idy * imgWidth + idx) * 4;
		if (idx < imgWidth && idy < imgHeight){
			float3 sum = make_float3(0.f,0.f,0.f);
			for (uint j = 0; j < matSize; j++) 
			{
				for (uint i = 0; i < matSize; i++) 
				{
					int dX = idx + i - matSize / 2;
					int dY = idy + j - matSize / 2;

					// Handle borders
					if (dX < 0) 
						dX = 0;

					if (dX >= imgWidth) 
						dX = imgWidth - 1;

					if (dY < 0) 
						dY = 0;

					if (dY >= imgHeight) 
						dY = imgHeight - 1;
					
					const int idMat = j * matSize + i;
					const int idPixel = (dY * imgWidth + dX) * 4;
					sum.x += (float)tex1Dfetch(texRef,idPixel) * dev_matConv[idMat];
					sum.y += (float)tex1Dfetch(texRef,idPixel+1) * dev_matConv[idMat];
					sum.z += (float)tex1Dfetch(texRef,idPixel+2) * dev_matConv[idMat];
				}
			}
			dev_output[index] = (uchar)max(0.f,min(255.f,sum.x));
			dev_output[index+1] = (uchar)max(0.f,min(255.f,sum.y));
			dev_output[index+2] = (uchar)max(0.f,min(255.f,sum.z));
			dev_output[index+3] = 255;
		}
	}

	__global__ void applyConvolutionv4(const uint imgWidth, const uint imgHeight,  const uint matSize, unsigned char* dev_output)
	{
		unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
		int index = (idy * imgWidth + idx) * 4;
		if (idx < imgWidth && idy < imgHeight){
			float3 sum = make_float3(0.f,0.f,0.f);
			for (uint j = 0; j < matSize; j++) 
			{
				for (uint i = 0; i < matSize; i++) 
				{
					int dX = idx + i - matSize / 2;
					int dY = idy + j - matSize / 2;

					// Handle borders
					if (dX < 0) 
						dX = 0;

					if (dX >= imgWidth) 
						dX = imgWidth - 1;

					if (dY < 0) 
						dY = 0;

					if (dY >= imgHeight) 
						dY = imgHeight - 1;
					
					const int idMat = j * matSize + i;
					const uchar c = tex2D(tex2DRef,dX,dY);
					sum.x += (float)(c) * dev_matConv[idMat];
					sum.y += (float)(c+1) * dev_matConv[idMat];
					sum.z += (float)(c+2) * dev_matConv[idMat];
				}
			}
			dev_output[index] = (uchar)max(0.f,min(255.f,sum.x));
			dev_output[index+1] = (uchar)max(0.f,min(255.f,sum.y));
			dev_output[index+2] = (uchar)max(0.f,min(255.f,sum.z));
			dev_output[index+3] = 255;
		}
	}

	
    void studentJob1(const std::vector<uchar4> &inputImg, // Input image
		const uint imgWidth, const uint imgHeight, // Image size
		const std::vector<float> &matConv, // Convolution matrix (square)
		const uint matSize, // Matrix size (width or height)
		const std::vector<uchar4> &resultCPU, // Just for comparison
		std::vector<uchar4> &output // Output image
		)
	{
		ChronoGPU chrGPU;

		// 3 arrays for GPU
		uchar* dev_input = NULL;
		uchar* dev_output = NULL;
		float* dev_matConv = NULL;
		chrGPU.start();
		dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight/threadsPerBlock.y));

		// allocate GPU buffers
		HANDLE_ERROR(hipMalloc((void**)&dev_input, imgWidth * imgHeight * 4 * sizeof(uchar)));
		HANDLE_ERROR(hipMalloc((void**)&dev_output, imgWidth * imgHeight * 4 * sizeof(uchar)));
		HANDLE_ERROR(hipMalloc((void**)&dev_matConv, matSize * matSize * sizeof(float)));

		// Copy data from host to device (input arrays) 
		HANDLE_ERROR(hipMemcpy(dev_input, inputImg.data(), (imgWidth * imgHeight * 4) * sizeof(uchar), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(dev_matConv, matConv.data(), matSize * matSize * sizeof(float), hipMemcpyHostToDevice));

		//launch kernel
		applyConvolution<<<numBlocks,threadsPerBlock>>>(dev_input,imgWidth,imgHeight,matSize,dev_matConv,dev_output);

		chrGPU.stop();
		std::cout << "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		HANDLE_ERROR(hipDeviceSynchronize());
		
		// Copy data from device to host (output array)
		HANDLE_ERROR(hipMemcpy(output.data(), dev_output, (imgWidth * imgHeight * 4) * sizeof(uchar), hipMemcpyDeviceToHost));

		// Free arrays on device
		HANDLE_ERROR(hipFree(dev_output));
		HANDLE_ERROR(hipFree(dev_matConv));
		HANDLE_ERROR(hipFree(dev_input));
	}

	void studentJob2(const std::vector<uchar4> &inputImg, // Input image
		const uint imgWidth, const uint imgHeight, // Image size
		const std::vector<float> &matConv, // Convolution matrix (square)
		const uint matSize, // Matrix size (width or height)
		const std::vector<uchar4> &resultCPU, // Just for comparison
		std::vector<uchar4> &output // Output image
		)
	{
		ChronoGPU chrGPU;

		// 3 arrays for GPU
		uchar* dev_input = NULL;
		uchar* dev_output = NULL;

		chrGPU.start();
		dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight/threadsPerBlock.y));

		// allocate GPU buffers
		HANDLE_ERROR(hipMalloc((void**)&dev_input, imgWidth * imgHeight * 4 * sizeof(uchar)));
		HANDLE_ERROR(hipMalloc((void**)&dev_output, imgWidth * imgHeight * 4 * sizeof(uchar)));

		// Copy data from host to device (input arrays) 
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_matConv), matConv.data(), matSize*matSize*sizeof(float)));
		HANDLE_ERROR(hipMemcpy(dev_input, inputImg.data(), (imgWidth * imgHeight * 4) * sizeof(uchar), hipMemcpyHostToDevice));

		//launch kernel
		applyConvolutionv2<<<numBlocks,threadsPerBlock>>>(dev_input,imgWidth,imgHeight,matSize,dev_output);

		chrGPU.stop();
		std::cout << "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		HANDLE_ERROR(hipDeviceSynchronize());
		
		// Copy data from device to host (output array)
		HANDLE_ERROR(hipMemcpy(output.data(), dev_output, (imgWidth * imgHeight * 4) * sizeof(uchar), hipMemcpyDeviceToHost));

		// Free arrays on device
		HANDLE_ERROR(hipFree(dev_output));
		HANDLE_ERROR(hipFree(dev_input));
	}

	void studentJob3(const std::vector<uchar4> &inputImg, // Input image
		const uint imgWidth, const uint imgHeight, // Image size
		const std::vector<float> &matConv, // Convolution matrix (square)
		const uint matSize, // Matrix size (width or height)
		const std::vector<uchar4> &resultCPU, // Just for comparison
		std::vector<uchar4> &output // Output image
		)
	{
		ChronoGPU chrGPU;

		// 3 arrays for GPU
		uchar* dev_input = NULL;
		uchar* dev_output = NULL;

		chrGPU.start();
		dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight/threadsPerBlock.y));

		// allocate GPU buffers
		HANDLE_ERROR(hipMalloc((void**)&dev_input, imgWidth * imgHeight * 4 * sizeof(uchar)));
		HANDLE_ERROR(hipMalloc((void**)&dev_output, imgWidth * imgHeight * 4 * sizeof(uchar)));

		// Copy data from host to device (input arrays) 
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_matConv), matConv.data(), matSize*matSize*sizeof(float)));

		// bind texture
		HANDLE_ERROR(hipBindTexture(NULL, texRef, dev_input,  imgWidth * imgHeight * 4 * sizeof(uchar)));

		//launch kernel
		applyConvolutionv3<<<numBlocks,threadsPerBlock>>>(imgWidth,imgHeight,matSize,dev_output);

		chrGPU.stop();
		std::cout << "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		HANDLE_ERROR(hipDeviceSynchronize());

		// Copy data from device to host (output array)
		HANDLE_ERROR(hipMemcpy(output.data(), dev_output, (imgWidth * imgHeight * 4) * sizeof(uchar), hipMemcpyDeviceToHost));

		// Free arrays on device
		HANDLE_ERROR(hipUnbindTexture(texRef));
		HANDLE_ERROR(hipFree(dev_output));
		HANDLE_ERROR(hipFree(dev_input));
	}

    void studentJob4(const std::vector<uchar4> &inputImg, // Input image
					const uint imgWidth, const uint imgHeight, // Image size
                    const std::vector<float> &matConv, // Convolution matrix (square)
					const uint matSize, // Matrix size (width or height)
					const std::vector<uchar4> &resultCPU, // Just for comparison
                    std::vector<uchar4> &output // Output image
					)
	{
		// 3 arrays for GPU
		uchar* dev_input = NULL;
		uchar* dev_output = NULL;
		size_t pitch;

		dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight/threadsPerBlock.y));

		hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar4>();
		hipArray *cuArray;

		HANDLE_ERROR(hipMallocArray(&cuArray,&channelDesc,imgWidth,imgHeight));
	
		// allocate GPU buffers
		HANDLE_ERROR(hipMallocPitch((void**)&dev_input, &pitch, (imgWidth * 4) * sizeof(uchar), imgHeight));
		HANDLE_ERROR(hipMallocPitch((void**)&dev_output, &pitch, (imgWidth * 4) * sizeof(uchar), imgHeight));

		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_matConv), matConv.data(), matSize*matSize*sizeof(float)));
		HANDLE_ERROR(hipMemcpyToArray(cuArray,0,0,inputImg.data(), imgWidth * imgHeight * 4 * sizeof(uchar),hipMemcpyHostToDevice));


		tex2DRef.addressMode[0] = hipAddressModeClamp;
		tex2DRef.addressMode[1] = hipAddressModeClamp;
		tex2DRef.normalized = false;
		
		// Copy data from host to device (input arrays) 
		HANDLE_ERROR(hipMemcpy2D(dev_input, pitch, inputImg.data(),  imgWidth * 4 * sizeof(uchar),  imgWidth * 4 * sizeof(uchar),  imgHeight, hipMemcpyHostToDevice));

		HANDLE_ERROR(hipBindTextureToArray(tex2DRef, cuArray, channelDesc));

		//launch kernel
		applyConvolutionv4<<<numBlocks,threadsPerBlock>>>(imgWidth,imgHeight,matSize,dev_output);

		HANDLE_ERROR(hipDeviceSynchronize());
 		// Copy data from device to host (output array)
		HANDLE_ERROR(hipMemcpy2D(output.data(), imgWidth * 4 * sizeof(uchar), dev_output, pitch, imgWidth * 4 * sizeof(uchar), imgHeight, hipMemcpyDeviceToHost));

		// Free arrays on device
		HANDLE_ERROR(hipFree(dev_output));
		HANDLE_ERROR(hipFreeArray(cuArray));
		HANDLE_ERROR(hipFree(dev_input));
		
		HANDLE_ERROR(hipUnbindTexture(tex2DRef));
	}

	void studentJob(const std::vector<uchar4> &inputImg, // Input image
		const uint imgWidth, const uint imgHeight, // Image size
		const std::vector<float> &matConv, // Convolution matrix (square)
		const uint matSize, // Matrix size (width or height)
		const std::vector<uchar4> &resultCPU, // Just for comparison
		std::vector<uchar4> &output // Output image
		)
	{
		std::cout << "Student Job 1 : " << std::endl;
		studentJob1(inputImg,imgWidth,imgHeight,matConv,matSize,resultCPU,output);
		std::cout << "Student Job 2 : " << std::endl;
		studentJob2(inputImg,imgWidth,imgHeight,matConv,matSize,resultCPU,output);
		std::cout << "Student Job 3 : " << std::endl;
		studentJob3(inputImg,imgWidth,imgHeight,matConv,matSize,resultCPU,output);
		std::cout << "Student Job 4 : " << std::endl;
		studentJob4(inputImg,imgWidth,imgHeight,matConv,matSize,resultCPU,output);
	}
}
