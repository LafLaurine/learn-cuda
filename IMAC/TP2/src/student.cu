#include "hip/hip_runtime.h"
/*
* TP 2 - Convolution d'images
* --------------------------
* Mémoire constante et textures
*
* File: student.cu
* Author: Maxime MARIA
*/

#include <hip/hip_runtime.h>
#include <>

#include <iostream>

#include "student.hpp"
#include "chronoGPU.hpp"

namespace IMAC
{

// ================================================== For image comparison
	std::ostream &operator <<(std::ostream &os, const uchar4 &c)
	{
		os << "[" << uint(c.x) << "," << uint(c.y) << "," << uint(c.z) << "," << uint(c.w) << "]";  
    	return os; 
	}

	void compareImages(const std::vector<uchar4> &a, const std::vector<uchar4> &b)
	{
		bool error = false;
		if (a.size() != b.size())
		{
			std::cout << "Size is different !" << std::endl;
			error = true;
		}
		else
		{
			for (uint i = 0; i < a.size(); ++i)
			{
				// Floating precision can cause small difference between host and device
				if (	std::abs(a[i].x - b[i].x) > 2 || std::abs(a[i].y - b[i].y) > 2 
					|| std::abs(a[i].z - b[i].z) > 2 || std::abs(a[i].w - b[i].w) > 2)
				{
					std::cout << "Error at index " << i << ": a = " << a[i] << " - b = " << b[i] << " - " << std::abs(a[i].x - b[i].x) << std::endl;
					error = true;
					break;
				}
			}
		}
		if (error)
		{
			std::cout << " -> You failed, retry!" << std::endl;
		}
		else
		{
			std::cout << " -> Well done!" << std::endl;
		}
	}
// ==================================================

	__global__ void convGPU(const uint imgWidth, const uint imgHeight, const uint matSize, uchar4* inputImg, float* matConv, uchar4* output)
	{
		uint x = (blockIdx.x * blockDim.x) + threadIdx.x;
		uint y = (blockIdx.y * blockDim.y) + threadIdx.y;
		uint idx = (y * imgWidth + x) * 3;
		if (idx < imgWidth * imgHeight * 3)
		{
			float3 sum = make_float3(0.f,0.f,0.f);
			// Apply convolution
			for ( uint j = 0; j < matSize; ++j ) 
			{
				for ( uint i = 0; i < matSize; ++i ) 
				{
					int dX = x + i - matSize / 2;
					int dY = y + j - matSize / 2;

					// Handle borders
					if ( dX < 0 ) 
						dX = 0;

					if ( dX >= imgWidth ) 
						dX = imgWidth - 1;

					if ( dY < 0 ) 
						dY = 0;

					if ( dY >= imgHeight ) 
						dY = imgHeight - 1;

					const int idMat		= j * matSize + i;
					const int idPixel	= dY * imgWidth + dX;
					sum.x += (float)inputImg[idPixel].x * matConv[idMat];
					sum.y += (float)inputImg[idPixel].y * matConv[idMat];
					sum.z += (float)inputImg[idPixel].z * matConv[idMat];
				}
			}
			//output[idx].x = (uchar)clampf( sum.x, 0.f, 255.f );
			//output[idx].y = (uchar)clampf( sum.y, 0.f, 255.f );
			//output[idx].z = (uchar)clampf( sum.z, 0.f, 255.f );
			//output[idx].w = 255;
		}
	}

    void studentJob(const std::vector<uchar4> &inputImg, // Input image
					const uint imgWidth, const uint imgHeight, // Image size
                    const std::vector<float> &matConv, // Convolution matrix (square)
					const uint matSize, // Matrix size (width or height)
					const std::vector<uchar4> &resultCPU, // Just for comparison
                    std::vector<uchar4> &output // Output image
					)
	{
		ChronoGPU chrGPU;

		// 3 arrays for GPU
		uchar4* d_inputImg = nullptr;
		float* d_matConv = nullptr;
		uchar4* d_output = nullptr;

		// Allocate arrays
		hipMalloc(&d_inputImg, sizeof(uchar4) * inputImg.size());
		hipMalloc(&d_matConv, sizeof(float) * matConv.size());
		hipMalloc(&d_output, sizeof(uchar4) * output.size()); // TODO check if output has the right size because might be 0

		// Copy data from host to device
		hipMemcpy(d_inputImg, inputImg.data(), sizeof(uchar4) * inputImg.size(), hipMemcpyHostToDevice);
		hipMemcpy(d_matConv, matConv.data(), sizeof(float) * matConv.size(), hipMemcpyHostToDevice);
		hipMemcpy(d_output, output.data(), sizeof(uchar4) * output.size(), hipMemcpyHostToDevice);

		// Launch kernel
		int blocks = (imgWidth * imgHeight) / 512;
		convGPU<<<blocks, 512>>>(imgWidth, imgHeight, matSize, d_inputImg, d_matConv, d_output);
		hipDeviceSynchronize();

		// Copy data from device to host (output array)
		hipMemcpy(output.data(), d_output, sizeof(uchar4) * output.size(), hipMemcpyDeviceToHost);

		// Free arrays on device
		hipFree(d_inputImg);
		hipFree(d_matConv);
		hipFree(d_output);
	}
}
