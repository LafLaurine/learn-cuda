#include "hip/hip_runtime.h"
/*
* TP 2 - Convolution d'images
* --------------------------
* Mémoire constante et textures
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"
#include "chronoGPU.hpp"
#include <algorithm>

#define BLOCK_SIZE 32
#define KERNEL_SIZE 16

namespace IMAC
{

// ================================================== For image comparison
	std::ostream &operator <<(std::ostream &os, const uchar4 &c)
	{
		os << "[" << uint(c.x) << "," << uint(c.y) << "," << uint(c.z) << "," << uint(c.w) << "]";  
    	return os; 
	}

	void compareImages(const std::vector<uchar4> &a, const std::vector<uchar4> &b)
	{
		bool error = false;
		if (a.size() != b.size())
		{
			std::cout << "Size is different !" << std::endl;
			error = true;
		}
		else
		{
			for (uint i = 0; i < a.size(); ++i)
			{
				// Floating precision can cause small difference between host and device
				if (std::abs(a[i].x - b[i].x) > 2 || std::abs(a[i].y - b[i].y) > 2 
					|| std::abs(a[i].z - b[i].z) > 2 || std::abs(a[i].w - b[i].w) > 2)
				{
					std::cout << "Error at index " << i << ": a = " << a[i] << " - b = " << b[i] << " - " << std::abs(a[i].x - b[i].x) << std::endl;
					error = true;
					break;
				}
			}
		}
		if (error)
		{
			std::cout << " -> You failed, retry!" << std::endl;
		}
		else
		{
			std::cout << " -> Well done!" << std::endl;
		}
	}

// ==================================================
	
	__global__ void applyConvolution(const unsigned char* dev_input, const uint imgWidth, const uint imgHeight, const uint matSize, float* dev_matConv, unsigned char* dev_output)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		int index = (idy * imgWidth + idx) * 4;
		if (idx < imgWidth && idy < imgHeight){
			float3 sum = make_float3(0.f,0.f,0.f);
			
			// Apply convolution
			for (uint j = 0; j < matSize; j++) 
			{
				for (uint i = 0; i < matSize; i++) 
				{
					int dX = idx + i - matSize / 2;
					int dY = idy + j - matSize / 2;

					// Handle borders
					if (dX < 0) 
						dX = 0;

					if (dX >= imgWidth) 
						dX = imgWidth - 1;

					if (dY < 0) 
						dY = 0;

					if (dY >= imgHeight) 
						dY = imgHeight - 1;
					
					const int idMat = j * matSize + i;
					const int idPixel = (dY * imgWidth + dX) * 4;
					sum.x += (float)dev_input[idPixel] * dev_matConv[idMat];
					sum.y += (float)dev_input[idPixel+1] * dev_matConv[idMat];
					sum.z += (float)dev_input[idPixel+2] * dev_matConv[idMat];
				}
			}
			dev_output[index] = (uchar)max(0.f,min(255.f,sum.x));
			dev_output[index+1] = (uchar)max(0.f,min(255.f,sum.y));
			dev_output[index+2] = (uchar)max(0.f,min(255.f,sum.z));
			dev_output[index+3] = 255;
		}
	}

	__device__ __constant__ float dev_matConv[KERNEL_SIZE * KERNEL_SIZE];

	__global__ void applyConvolutionv2(const unsigned char* dev_input, const uint imgWidth, const uint imgHeight,  const uint matSize, unsigned char* dev_output)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		int idy = blockIdx.y * blockDim.y + threadIdx.y;
		int index = (idy * imgWidth + idx) * 4;
		if (idx < imgWidth && idy < imgHeight){
			float3 sum = make_float3(0.f,0.f,0.f);
			for (uint j = 0; j < matSize; j++) 
			{
				for (uint i = 0; i < matSize; i++) 
				{
					int dX = idx + i - matSize / 2;
					int dY = idy + j - matSize / 2;

					// Handle borders
					if (dX < 0) 
						dX = 0;

					if (dX >= imgWidth) 
						dX = imgWidth - 1;

					if (dY < 0) 
						dY = 0;

					if (dY >= imgHeight) 
						dY = imgHeight - 1;
					
					const int idMat = j * matSize + i;
					const int idPixel = (dY * imgWidth + dX) * 4;
					sum.x += (float)dev_input[idPixel] * dev_matConv[idMat];
					sum.y += (float)dev_input[idPixel+1] * dev_matConv[idMat];
					sum.z += (float)dev_input[idPixel+2] * dev_matConv[idMat];
				}
			}
			dev_output[index] = (uchar)max(0.f,min(255.f,sum.x));
			dev_output[index+1] = (uchar)max(0.f,min(255.f,sum.y));
			dev_output[index+2] = (uchar)max(0.f,min(255.f,sum.z));
			dev_output[index+3] = 255;
		}
	}

	texture<uchar> texRef;
	texture<uchar4,2> tex2DRef;

	__global__ void applyConvolutionv3(const uint imgWidth, const uint imgHeight,  const uint matSize, unsigned char* dev_output)
	{
		unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
		int index = (idy * imgWidth + idx) * 4;
		if (idx < imgWidth && idy < imgHeight){
			float3 sum = make_float3(0.f,0.f,0.f);
			for (uint j = 0; j < matSize; j++) 
			{
				for (uint i = 0; i < matSize; i++) 
				{
					int dX = idx + i - matSize / 2;
					int dY = idy + j - matSize / 2;

					// Handle borders
					if (dX < 0) 
						dX = 0;

					if (dX >= imgWidth) 
						dX = imgWidth - 1;

					if (dY < 0) 
						dY = 0;

					if (dY >= imgHeight) 
						dY = imgHeight - 1;
					
					const int idMat = j * matSize + i;
					const int idPixel = (dY * imgWidth + dX) * 4;
					sum.x += (float)tex1Dfetch(texRef,idPixel) * dev_matConv[idMat];
					sum.y += (float)tex1Dfetch(texRef,idPixel+1) * dev_matConv[idMat];
					sum.z += (float)tex1Dfetch(texRef,idPixel+2) * dev_matConv[idMat];
				}
			}
			dev_output[index] = (uchar)max(0.f,min(255.f,sum.x));
			dev_output[index+1] = (uchar)max(0.f,min(255.f,sum.y));
			dev_output[index+2] = (uchar)max(0.f,min(255.f,sum.z));
			dev_output[index+3] = 255.f;
		}
	}

	__global__ void applyConvolutionv4(const uint imgWidth, const uint imgHeight,  const uint matSize, unsigned char* dev_output)
	{
		unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
		unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
		int index = (idy * imgWidth + idx) * 4;
		if (idx < imgWidth && idy < imgHeight){
			float4 sum = make_float4(0.f,0.f,0.f,0.f);
			for (uint j = 0; j < matSize; j++) 
			{
				for (uint i = 0; i < matSize; i++) 
				{
					int dX = idx + i - matSize / 2;
					int dY = idy + j - matSize / 2;

					// Handle borders
					if (dX < 0) 
						dX = 0;

					if (dX >= imgWidth) 
						dX = imgWidth - 1;

					if (dY < 0) 
						dY = 0;

					if (dY >= imgHeight) 
						dY = imgHeight - 1;
					
					const int idMat = j * matSize + i;
					const uchar4 c = tex2D(tex2DRef,dX,dY);
					sum.x += (float)(c.x) * dev_matConv[idMat];
					sum.y += (float)(c.y) * dev_matConv[idMat];
					sum.z += (float)(c.z) * dev_matConv[idMat];
				}
			}
			dev_output[index] = (uchar)max(0.f,min(255.f,sum.x));
			dev_output[index+1] = (uchar)max(0.f,min(255.f,sum.y));
			dev_output[index+2] = (uchar)max(0.f,min(255.f,sum.z));
			dev_output[index+3] = 255.f;
		}
	}

    void studentJob1(const std::vector<uchar4> &inputImg, // Input image
		const uint imgWidth, const uint imgHeight, // Image size
		const std::vector<float> &matConv, // Convolution matrix (square)
		const uint matSize, // Matrix size (width or height)
		std::vector<uchar4> &output // Output image
		)
	{
		ChronoGPU chrGPU;

		// 3 arrays for GPU
		uchar* dev_input = NULL;
		uchar* dev_output = NULL;
		float* dev_matConv = NULL;
		chrGPU.start();
		dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight/threadsPerBlock.y));

		// allocate GPU buffers
		HANDLE_ERROR(hipMalloc((void**)&dev_input, imgWidth * imgHeight * 4 * sizeof(uchar)));
		HANDLE_ERROR(hipMalloc((void**)&dev_output, imgWidth * imgHeight * 4 * sizeof(uchar)));
		HANDLE_ERROR(hipMalloc((void**)&dev_matConv, matSize * matSize * sizeof(float)));

		// Copy data from host to device (input arrays) 
		HANDLE_ERROR(hipMemcpy(dev_input, inputImg.data(), (imgWidth * imgHeight * 4) * sizeof(uchar), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(dev_matConv, matConv.data(), matSize * matSize * sizeof(float), hipMemcpyHostToDevice));

		//launch kernel
		applyConvolution<<<numBlocks,threadsPerBlock>>>(dev_input,imgWidth,imgHeight,matSize,dev_matConv,dev_output);

		chrGPU.stop();
		std::cout << "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		HANDLE_ERROR(hipDeviceSynchronize());
		
		// Copy data from device to host (output array)
		HANDLE_ERROR(hipMemcpy(output.data(), dev_output, (imgWidth * imgHeight * 4) * sizeof(uchar), hipMemcpyDeviceToHost));

		// Free arrays on device
		HANDLE_ERROR(hipFree(dev_output));
		HANDLE_ERROR(hipFree(dev_matConv));
		HANDLE_ERROR(hipFree(dev_input));
	}

	void studentJob2(const std::vector<uchar4> &inputImg, // Input image
		const uint imgWidth, const uint imgHeight, // Image size
		const std::vector<float> &matConv, // Convolution matrix (square)
		const uint matSize, // Matrix size (width or height)
		std::vector<uchar4> &output // Output image
		)
	{
		ChronoGPU chrGPU;

		// 3 arrays for GPU
		uchar* dev_input = NULL;
		uchar* dev_output = NULL;

		chrGPU.start();
		dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight/threadsPerBlock.y));

		// allocate GPU buffers
		HANDLE_ERROR(hipMalloc((void**)&dev_input, imgWidth * imgHeight * 4 * sizeof(uchar)));
		HANDLE_ERROR(hipMalloc((void**)&dev_output, imgWidth * imgHeight * 4 * sizeof(uchar)));

		// Copy data from host to device (input arrays) 
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_matConv), matConv.data(), matSize*matSize*sizeof(float)));
		HANDLE_ERROR(hipMemcpy(dev_input, inputImg.data(), (imgWidth * imgHeight * 4) * sizeof(uchar), hipMemcpyHostToDevice));

		//launch kernel
		applyConvolutionv2<<<numBlocks,threadsPerBlock>>>(dev_input,imgWidth,imgHeight,matSize,dev_output);

		chrGPU.stop();
		std::cout << "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		HANDLE_ERROR(hipDeviceSynchronize());
		
		// Copy data from device to host (output array)
		HANDLE_ERROR(hipMemcpy(output.data(), dev_output, (imgWidth * imgHeight * 4) * sizeof(uchar), hipMemcpyDeviceToHost));

		// Free arrays on device
		HANDLE_ERROR(hipFree(dev_output));
		HANDLE_ERROR(hipFree(dev_input));
	}

	void studentJob3(const std::vector<uchar4> &inputImg, // Input image
		const uint imgWidth, const uint imgHeight, // Image size
		const std::vector<float> &matConv, // Convolution matrix (square)
		const uint matSize, // Matrix size (width or height)
		std::vector<uchar4> &output // Output image
		)
	{
		ChronoGPU chrGPU;

		// 3 arrays for GPU
		uchar* dev_input = NULL;
		uchar* dev_output = NULL;

		chrGPU.start();
		dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight/threadsPerBlock.y));

		// allocate GPU buffers
		HANDLE_ERROR(hipMalloc((void**)&dev_input, imgWidth * imgHeight * 4 * sizeof(uchar)));
		HANDLE_ERROR(hipMalloc((void**)&dev_output, imgWidth * imgHeight * 4 * sizeof(uchar)));

		// Copy data from host to device (input arrays) 
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_matConv), matConv.data(), matSize*matSize*sizeof(float)));
		HANDLE_ERROR(hipMemcpy(dev_input, inputImg.data(), (imgWidth * imgHeight * 4) * sizeof(uchar), hipMemcpyHostToDevice));
		// bind texture
		HANDLE_ERROR(hipBindTexture(NULL, texRef, dev_input,  imgWidth * imgHeight * 4 * sizeof(uchar)));

		//launch kernel
		applyConvolutionv3<<<numBlocks,threadsPerBlock>>>(imgWidth,imgHeight,matSize,dev_output);

		chrGPU.stop();
		std::cout << "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		HANDLE_ERROR(hipDeviceSynchronize());

		// Copy data from device to host (output array)
		HANDLE_ERROR(hipMemcpy(output.data(), dev_output, (imgWidth * imgHeight * 4) * sizeof(uchar), hipMemcpyDeviceToHost));

		// Free arrays on device
		HANDLE_ERROR(hipUnbindTexture(texRef));
		HANDLE_ERROR(hipFree(dev_output));
		HANDLE_ERROR(hipFree(dev_input));
	}

    void studentJob4(const std::vector<uchar4> &inputImg, // Input image
					const uint imgWidth, const uint imgHeight, // Image size
                    const std::vector<float> &matConv, // Convolution matrix (square)
					const uint matSize, // Matrix size (width or height)
                    std::vector<uchar4> &output // Output image
					)
	{
		// 3 arrays for GPU
		uchar* dev_input = NULL;
		uchar* dev_output = NULL;
		size_t pitch;

		dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 numBlocks(ceil((float)imgWidth / threadsPerBlock.x), ceil((float)imgHeight/threadsPerBlock.y));

		// allocate GPU buffers
		HANDLE_ERROR(hipMallocPitch((void**)&dev_input, &pitch, (imgWidth) * sizeof(uchar) * 4, imgHeight));

		HANDLE_ERROR(hipMemcpy2D(dev_input,pitch,inputImg.data(),  sizeof(uchar) * 4 *imgWidth,imgWidth * sizeof(uchar) * 4, imgHeight, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_matConv), matConv.data(), matSize*matSize*sizeof(float)));
		
		HANDLE_ERROR(hipBindTexture2D(NULL, tex2DRef, dev_input, tex2DRef.channelDesc, imgWidth, imgHeight, pitch));

		//tex2DRef.normalized = false;  // don't use normalized values                   

		HANDLE_ERROR(hipMalloc(&dev_output, 4*imgWidth*imgHeight*sizeof(uchar)));

		//launch kernel
		applyConvolutionv4<<<numBlocks,threadsPerBlock>>>(imgWidth,imgHeight,matSize,dev_output);

		HANDLE_ERROR(hipDeviceSynchronize());
 		// Copy data from device to host (output array)
		HANDLE_ERROR(hipMemcpy(output.data(), dev_output, 4 * imgHeight * imgWidth * sizeof(uchar), hipMemcpyDeviceToHost));

		HANDLE_ERROR(hipUnbindTexture(tex2DRef));
		// Free arrays on device
		HANDLE_ERROR(hipFree(dev_output));
		HANDLE_ERROR(hipFree(dev_input));
	}

	void studentJob(const std::vector<uchar4> &inputImg, // Input image
		const uint imgWidth, const uint imgHeight, // Image size
		const std::vector<float> &matConv, // Convolution matrix (square)
		const uint matSize, // Matrix size (width or height)
		const std::vector<uchar4> &resultCPU, // Just for comparison
		std::vector<uchar4> &output // Output image
		)
	{
		std::cout << "------------------" << std::endl;
		std::cout << "GPU " << std::endl;
		std::cout << "Student Job 1 : " << std::endl;
		studentJob1(inputImg,imgWidth,imgHeight,matConv,matSize,output);
		compareImages(resultCPU,output);
		std::cout << "Student Job 2 : " << std::endl;
		studentJob2(inputImg,imgWidth,imgHeight,matConv,matSize,output);
		compareImages(resultCPU,output);

		std::cout << "Student Job 3 : " << std::endl;
		studentJob3(inputImg,imgWidth,imgHeight,matConv,matSize,output);
		compareImages(resultCPU,output);
		std::cout << "Student Job 4 : " << std::endl;
		studentJob4(inputImg,imgWidth,imgHeight,matConv,matSize,output);
		compareImages(resultCPU,output);
	}
}
