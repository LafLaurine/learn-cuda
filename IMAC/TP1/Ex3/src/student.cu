#include "hip/hip_runtime.h"
/*
* TP 1 - Premiers pas en CUDA
* --------------------------
* Ex 3: Filtre d'images sepia
*
* File: student.cu
* Author: Maxime MARIA
*/

#include "student.hpp"
#include "chronoGPU.hpp"

namespace IMAC
{
	__global__ void sepiaImageCUDA(const int width, const int height, const unsigned char* dev_input, unsigned char* dev_output)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		dev_output[idx] = dev_input[idx];
	}

	void studentJob(const std::vector<uchar> &input, const uint width, const uint height, std::vector<uchar> &output)
	{
		ChronoGPU chrGPU;

		// 2 arrays for GPU
		uchar *dev_input = NULL;
		uchar *dev_output = NULL;

		// Allocate memory
		const size_t bytes = width * height * 3 * sizeof(uchar);
		hipMalloc(&dev_input, bytes);
		hipMalloc(&dev_output, bytes);
		
		// Copy data from host to device
		hipMemcpy(dev_input, input.data(), bytes, hipMemcpyHostToDevice);

		// Launch kernel
		sepiaImageCUDA<<<1, 256>>>(width, height, dev_input, dev_output);

		// Copy data from device to host
		hipMemcpy(output.data(), dev_output, bytes, hipMemcpyDeviceToHost);

		// Free memory
		hipFree(dev_input);
		hipFree(dev_output);
	}
}
