#include "hip/hip_runtime.h"
/*
* TP 1 - Premiers pas en CUDA
* --------------------------
* Ex 2: Addition de vecteurs
*
* File: student.cu
* Author: Maxime MARIA
*/

#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <iostream>

#include "student.hpp"
#include "chronoGPU.hpp"

namespace IMAC
{
	__global__ void sumArraysCUDA(const int n, const int *const dev_a, const int *const dev_b, int *const dev_res)
	{
		dev_res[threadIdx.x] = dev_a[threadIdx.x] + dev_b[threadIdx.x];
	}

    void studentJob(const int size, const int *const a, const int *const b, int *const res)
	{
		ChronoGPU chrGPU;

		// 3 arrays for GPU
		int *dev_a = NULL;
		int *dev_b = NULL;
		int *dev_res = NULL;

		// Allocate arrays on device (input and ouput)
		const size_t bytes = size * sizeof(int);
		std::cout 	<< "Allocating input (3 arrays): " 
					<< ( ( 3 * bytes ) >> 20 ) << " MB on Device" << std::endl;
		chrGPU.start();
		
		hipMalloc(&dev_a, bytes);
		hipMalloc(&dev_b, bytes);
		hipMalloc(&dev_res, bytes);
		
		chrGPU.stop();
		std::cout 	<< "-> Done : " << chrGPU.elapsedTime() << " ms" << std::endl << std::endl;

		// Copy data from host to device (input arrays) 
		hipMemcpy(dev_a, a, bytes, hipMemcpyHostToDevice);
		hipMemcpy(dev_b, b, bytes, hipMemcpyHostToDevice);

		// Launch kernel
		sumArraysCUDA<<<1, 256>>>(size, dev_a, dev_b, dev_res);

		// Copy data from device to host (output array)  
		hipMemcpy(res, dev_res, bytes, hipMemcpyDeviceToHost);

		// Free arrays on device
		hipFree(dev_a);
		hipFree(dev_b);
		hipFree(dev_res);
	}
}

