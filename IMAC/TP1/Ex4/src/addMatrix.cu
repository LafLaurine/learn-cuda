#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

#include <iostream>

#include "chronoGPU.hpp"
#include "common.hpp"

#define WIDTH 2
#define HEIGHT 2
#define SIZE HEIGHT*WIDTH

void createMatrix(int m[SIZE])
{
    for (int i = 0; i < SIZE; i++) {
        m[i] = rand()%100;
    }
}

void matricesSum(const int m1[SIZE], const int m2[SIZE])
{   
    int s[SIZE] = {0};
    std::cout << "Output of the sum : " << std::endl;

    for (int ix = 0; ix < WIDTH; ix++) { 
        for (int iy = 0; iy < HEIGHT; iy++) {
            s[iy*WIDTH + ix] = m1[iy*WIDTH + ix] + m2[iy*WIDTH + ix];
            std::cout << s[iy*WIDTH + ix] << std::endl;
        }
    }
}

__global__ void matSum(const int dev_a[SIZE], const int dev_b[SIZE], int dev_res[SIZE]){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = (idy * WIDTH + idx);
    if (idx < WIDTH && idy < HEIGHT){
        dev_res[index] = dev_a[index] + dev_b[index];
    }
}

void CPUCompute(const int m1[SIZE], const int m2[SIZE]) {
    matricesSum(m1,m2);
}

void GPUCompute(const int a[SIZE], const int b[SIZE], int res[SIZE]) {
    const int* dev_a[SIZE];
    const int* dev_b[SIZE];
    int* dev_res[SIZE];

    // Allocate arrays on device (input and ouput)
    const size_t bytes = SIZE*sizeof(int);

    hipMalloc((void**)&dev_a, bytes);
    hipMalloc((void**)&dev_b, bytes);
    hipMalloc((void**)&dev_res, bytes);

    //The error lays here
    hipMemcpy(dev_a, &a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, &b, bytes, hipMemcpyHostToDevice);
        
    int numBlocks = 1;
    dim3 threadsPerBlock(WIDTH*HEIGHT,HEIGHT*WIDTH);
    matSum<<<numBlocks, threadsPerBlock>>>(*dev_a,*dev_b,*dev_res);
    hipDeviceSynchronize();

    hipMemcpy(&res, dev_res, bytes, hipMemcpyDeviceToHost);

    std::cout << *res << std::endl;

    hipFree(dev_res);
    hipFree(dev_b);
    hipFree(dev_a);

}

int main()
{   
    int m1[SIZE];
    createMatrix(m1);
    int m2[SIZE];
    createMatrix(m2);
    int mres[SIZE];

    CPUCompute(m1,m2);
    GPUCompute(m1,m2,mres);
	return 0;
}

