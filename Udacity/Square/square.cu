
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

__global__ void square(float* d_out, float* d_in) 
{
    int idx = threadIdx.x;
    d_out[idx] = d_in[idx] * d_in[idx];
}

int main() 
{
    const int ARRAY_SIZE = 64;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    //generate input array on host
    float h_in[ARRAY_SIZE];
    for(int i = 0; i < ARRAY_SIZE; i++) {
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    //declare GPU memory pointers
    float* d_in;
    float* d_out;

    // allocate GPU memory
    hipMalloc((void**) &d_in, ARRAY_BYTES);
    hipMalloc((void**) &d_out, ARRAY_BYTES);

    // transfer the array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // launch the kernel
    square<<<1, ARRAY_SIZE>>>(d_out, d_in);

    // copy back the result array to the CPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);
    
    //print out the resulting array
    for(int i = 0; i < ARRAY_SIZE; i++) {
        std::cout << h_out[i] << std::endl;
    }

    //we don't forget to free the GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}