
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512


void random_ints(int* a, int n)
{
   int i;
   for (i = 0; i < n; ++i) {
       a[i] = rand() %5000;
   }
}

// a block can be split into parallel threads
__global__ void add(int *a, int *b, int *c)
{
    // use threadIdx.x to access thread index
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(void)
{
    // host copies of a, b, c
    int *a, *b, *c; 
    // device copies of a, b, c
    int *d_a, *d_b, *d_c; 
    int size = N * sizeof(int);

    // we need to allocate memory on the GPU
    // allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // allocate space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); 
    random_ints(a, N);
    b = (int *)malloc(size); 
    random_ints(b, N);
    c = (int *)malloc(size);

    // copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // launch add() kernel on the GPU with N threads
    add<<<1,N>>>(d_a, d_b, d_c);

    // copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // don't forget to free the memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // check error 
    printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));

    return 0;
}